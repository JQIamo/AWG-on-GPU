#include "spcm_cuda_common.h"

#include <cstdio>

// ----- CUDA include -----
#   include <hip/hip_runtime.h>

// CUDA-C includes
#   include <hip/hip_runtime.h>


// ----- Init CUDA device without RDMA support (e.g. on Windows). -----
// -- lCUDADevIdx:  index of CUDA device to be used
// -- return:       true if initialization succeeded, false otherwise
bool bInitCUDADevice (int lCUDADevIdx)
    {
    // ----- check for CUDA-capable devices -----
    int lCUDADeviceCount = 0;
    hipError_t eCudaErr = hipGetDeviceCount (&lCUDADeviceCount);
    if (eCudaErr != hipSuccess)
        {
        printf ("ERROR in hipGetDeviceCount(): %s\n", hipGetErrorString(eCudaErr));
        return false;
        }

    if (lCUDADeviceCount == 0)
        {
        printf ("ERROR: there are no available devices that support CUDA\n");
        return false;
        }
    if (lCUDADevIdx >= lCUDADeviceCount)
        {
        printf ("ERROR: requested device %d, but only %d CUDA device(s) available\n", lCUDADevIdx, lCUDADeviceCount);
        return false;
        }

    printf ("Detected %d CUDA Capable device(s).\n", lCUDADeviceCount);
    hipSetDevice (lCUDADevIdx);
    hipDeviceProp_t stCUDADeviceProp;
    hipGetDeviceProperties (&stCUDADeviceProp, lCUDADevIdx);

    printf("\nUsing device %d: \"%s\"\n", 0, stCUDADeviceProp.name);

    return true;
    }

#ifndef WIN32

// ----- Sets the CUDA device (GPU) to be used and allocates a buffer that's usable for RDMA. -----
// -- lCUDADevIdx:           index of CUDA device to be used
// -- qwDMABufferSize_bytes: size of the buffer that should be allocated
// -- return:                pointer to buffer of requested size if initialization succeeded, NULL otherwise
void* pvGetRDMABuffer (int lCUDADevIdx, size_t qwDMABufferSize_bytes)
    {
    // ----- check for CUDA-capable devices -----
    int lCUDADeviceCount = 0;
    hipError_t eCudaErr = hipGetDeviceCount (&lCUDADeviceCount);
    if (eCudaErr != hipSuccess)
        {
        printf ("ERROR in hipGetDeviceCount(): %s\n", hipGetErrorString(eCudaErr));
        return NULL;
        }

    if (lCUDADeviceCount == 0)
        {
        printf ("ERROR: there are no available devices that support CUDA\n");
        return NULL;
        }
    if (lCUDADevIdx >= lCUDADeviceCount)
        {
        printf ("ERROR: requested device %d, but only %d CUDA device(s) available\n", lCUDADevIdx, lCUDADeviceCount);
        return NULL;
        }

    printf ("Detected %d CUDA Capable device(s).\n", lCUDADeviceCount);
    hipSetDevice (lCUDADevIdx);
    hipDeviceProp_t stCUDADeviceProp;
    hipGetDeviceProperties (&stCUDADeviceProp, lCUDADevIdx);
    printf("\nUsing device %d: \"%s\"\n", 0, stCUDADeviceProp.name);

    // ----- we require at least CUDA 5.0 -----
    if (stCUDADeviceProp.major < 5)
        {
        printf ("ERROR: RDMA requires at least CUDA compute capability 5.0 (found: %d.%d)\n", stCUDADeviceProp.major, stCUDADeviceProp.minor);
        return NULL;
        }

    int lRDMASupported = 0;
    hipDeviceGetAttribute (&lRDMASupported, cudaDevAttrGPUDirectRDMASupported, lCUDADevIdx);
    if (lRDMASupported == 0)
        {
        printf ("ERROR: device \"%s\" does not support RDMA", stCUDADeviceProp.name);
        return NULL;
        }

    // ----- allocate DMA buffer on GPU -----
    void* pvDMABuffer_gpu = NULL;
    if (strncmp (stCUDADeviceProp.name, "Xavier", 6) == 0)
        eCudaErr = hipHostAlloc (&pvDMABuffer_gpu, qwDMABufferSize_bytes, hipHostMallocDefault); // required to use hipHostAlloc on Jetson
    else
        {
        eCudaErr = hipMalloc (&pvDMABuffer_gpu, qwDMABufferSize_bytes);
        }
    if (eCudaErr != hipSuccess)
        {
        printf ("ERROR in hipMalloc(): %s\n", hipGetErrorString(eCudaErr));
        return NULL;
        }

    // in GPUDirect RDMA scope should always be HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS
    unsigned int dwFlag = 1;
    hipError_t eResult = hipPointerSetAttribute (&dwFlag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t)pvDMABuffer_gpu);
    if (eResult != hipSuccess)
        {
        const char* szError;
        hipDrvGetErrorString (eResult, &szError);
        printf ("ERROR in hipPointerSetAttribute(ATTRIBUTE_SYNC_MEMOPS): %s\n", szError);
        hipFree (pvDMABuffer_gpu);
        return NULL;
        }

    return pvDMABuffer_gpu;
    }

#endif

// ----- Returns error description for CUDA FFT error code -----
const char* szCudaGetErrorText (hipfftResult eError)
    {
    switch (eError)
        {
        case HIPFFT_SUCCESS:         return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:    return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:    return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:    return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE:   return "HIPFFT_INVALID_VALUE";
        case HIPFFT_INTERNAL_ERROR:  return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:     return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:    return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:    return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA:  return "HIPFFT_UNALIGNED_DATA";
        }

    return "<unknown>";
    }
